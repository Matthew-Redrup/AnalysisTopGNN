#include "hip/hip_runtime.h"
#include "NuSol.cu"

template <typename scalar_t>
__global__ void _baseValsK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> muP2, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> bP2,
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mue, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> be,
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> cos,
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> sin, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mT2, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mW2, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mNu2,
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _out, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _bB, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _muB, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _muB2, 
		const int x, const int y)
{
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y;

	if (indx >= x || indy >= y){return;}
	if (indy == 0){ _out[indx][indy] = cos[indx][0]; return; }
	if (indy == 1){ _out[indx][indy] = sin[indx][0]; return; }
	if (indy == 2){ _out[indx][indy] = _x0(mW2[indx][0], mNu2[indx][0], muP2[indx][0], mue[indx][0]); return; }
	if (indy == 3){ _out[indx][indy] = _x0(mT2[indx][0], mW2[indx][0], bP2[indx][0], be[indx][0]); return; }
	if (indy == 4){ _bB[indx][0] = sqrt(_beta2(bP2[indx][0], be[indx][0])); return; }
	if (indy == 5){ _muB[indx][0] = sqrt(_beta2(muP2[indx][0], mue[indx][0])); return; }
	if (indy == 6)
	{
		_out[indx][indy] = _w(muP2[indx][0], bP2[indx][0], mue[indx][0], be[indx][0], cos[indx][0], sin[indx][0], 1); 
		return; 
	}
	if (indy == 7)
	{
		_out[indx][indy] = _w(muP2[indx][0], bP2[indx][0], mue[indx][0], be[indx][0], cos[indx][0], sin[indx][0], -1); 
		return; 
	}
	if (indy == 8){ _muB2[indx][0] = _beta2(muP2[indx][0], mue[indx][0]); return; }
	if (indy == 9){ _out[indx][12] = (mW2[indx][0] - mNu2[indx][0]); return; }


}

template <typename scalar_t>
__global__ void _baseValsK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> muB2,
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> muB,
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> muP2, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _out, 
		const int x, const int y)
{
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y;

	if (indx >= x || indy >= y){return;}
	if (indy == 0){ _out[indx][11] = _out[indx][6]*_out[indx][6] + 1 - muB2[indx][0]; return; } // O2
	if (indy == 1){ _out[indx][12] = _out[indx][12]*(1 - muB2[indx][0]); return; } // e2
	if (indy == 2){ _out[indx][4] = (_out[indx][2] * muB[indx][0] - sqrt(muP2[indx][0]) * ( 1 - muB2[indx][0] )) / muB2[indx][0]; return; } //Sx
}

template <typename scalar_t>
__global__ void _baseVals_K(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> bB, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> cos, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> sin, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _out, 
		const int x)
{
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 

	if (indx >= x){return;}
	_out[indx][5] = ((_out[indx][3] / bB[indx][0]) - cos[indx][0] * _out[indx][4]) / sin[indx][0]; return; 
}


template <typename scalar_t>
__global__ void _baseValsK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mW2,
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _out, 
		const int x, const int y)
{
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y;

	if (indx >= x || indy >= y){return;}
	if (indy == 0)
	{
		_out[indx][8] = _out[indx][4] - (_out[indx][4] + _out[indx][6]*_out[indx][5])/_out[indx][11]; // x1
		return; 
	}

	if (indy == 1)
	{
		_out[indx][9] = _out[indx][5] - (_out[indx][4] + _out[indx][6]*_out[indx][5])*(_out[indx][6]/_out[indx][11]); // y1
		return; 
	}
	if (indy == 2)
	{
		// Z_tmp = - (Sy - w*Sx)^2 - (mW2 - x0^2 - e2)
		_out[indx][10] = -(_out[indx][5] - _out[indx][6]*_out[indx][4])*(_out[indx][5] - _out[indx][6]*_out[indx][4]) 
			         - (mW2[indx][0] - (_out[indx][2]*_out[indx][2]) - _out[indx][12]); return; 
	}

}

template <typename scalar_t>
__global__ void _baseValsK(
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _out, 
		const int x)
{
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 
	if (indx >= x){return;}
	_out[indx][10] = _sqrt(_out[indx][8]*_out[indx][8]*_out[indx][11] + _out[indx][10]); 
}

template <typename scalar_t>
__global__ void _HMatrix(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> sols_,
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> muP_,
		torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
		const int x, const int y, const int z)
{
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y;
	const int indz = blockIdx.z;
	
	if (indx >= x || indy >= y || indz >= z){return;}
	if (indy == 0 || indz == 1){ return; }
	if (indy == 1 || indz == 1){ return; }
	if (indy == 2 || indz == 0){ return; }
	if (indy == 2 || indz == 2){ return; }

	if (indy == 0 || indz == 0){ out[indx][indy][indz] = sols_[indx][10]/_sqrt(sols_[indx][11]); return; }
	if (indy == 0 || indz == 2){ out[indx][indy][indz] = sols_[indx][8] - muP_[indx][0]; return; }
	if (indy == 1 || indz == 0){ out[indx][indy][indz] = (sols_[indx][10]/_sqrt(sols_[indx][11]))*sols_[indx][6]; return; }
	if (indy == 1 || indz == 2){ out[indx][indy][indz] = sols_[indx][9]; return; }
	if (indy == 2 || indz == 1){ out[indx][indy][indz] = sols_[indx][10]; return; }
}
