#include "hip/hip_runtime.h"
#include "Operators.cu"

template <typename scalar_t> 
__global__ void _Dot2K(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> v1, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> v2, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _out, 
		const int len, const int dim)
{
	
	const int indx = blockIdx.x*blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y; 
	
	if (indx >= len || indy >= dim){return;}
	_v1xv2(&_out[indx][indy], &v1[indx][indy], &v2[indx][indy]); 
}

template <typename scalar_t> 
__global__ void _CosThetaK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _v12, 
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _v22, 
		torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> _V1V2, 
		const int x)
{	
	const int indx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (indx >= x){return;}
	_costheta(&_V1V2[indx][0], &_v12[indx][0], &_v22[indx][0], &_V1V2[indx][0]); 
}

template <typename scalar_t>
__global__ void _RxK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> agl, 
		torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> _out)
{
	const int indx = blockIdx.x * blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y; 
	const int indz = blockIdx.z; 

	if (indx >= agl.size(0) || indy >= 3 || indz >= 3){ return; }
	
	if (indy == indz && indz == 0){ _out[indx][indy][indz] = 1; return; }	
	if (indy == indz && indz > 0){ _out[indx][indy][indz] = _cos(agl[indx][0]); return; }
	if (indy == 1 && indz == 2){ _out[indx][indy][indz] = -_sin(agl[indx][0]); return; }
	if (indy == 2 && indz == 1){ _out[indx][indy][indz] = _sin(agl[indx][0]); return; }
}

template <typename scalar_t>
__global__ void _RyK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> agl, 
		torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> _out)
{
	const int indx = blockIdx.x * blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y; 
	const int indz = blockIdx.z; 

	if (indx >= agl.size(0) || indy >= 3 || indz >= 3){ return; }

	if (indy == indz && ( indz == 0 || indz == 2) ){ _out[indx][indy][indz] = _cos(agl[indx][0]); return; }	
	if (indy == 0 && indz == 2){ _out[indx][indy][indz] = _sin(agl[indx][0]); return; }	
	if (indy == 2 && indz == 0){ _out[indx][indy][indz] = -_sin(agl[indx][0]); return; }	
	if (indy == indz && indz == 1){ _out[indx][indy][indz] = 1; return; }	
}

template <typename scalar_t>
__global__ void _RzK(
		const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> agl, 
		torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> _out)
{
	const int indx = blockIdx.x * blockDim.x + threadIdx.x; 
	const int indy = blockIdx.y; 
	const int indz = blockIdx.z; 

	if (indx >= agl.size(0) || indy >= 3 || indz >= 3){ return; }

	if (indy == indz && indz < 2){ _out[indx][indy][indz] = _cos(agl[indx][0]); return; }	
	if (indy == 1 && indz == 0){ _out[indx][indy][indz] = _sin(agl[indx][0]); return; }
	if (indy == 0 && indz == 1){ _out[indx][indy][indz] = -_sin(agl[indx][0]); return; }
	if (indy == indz && indz == 2){ _out[indx][indy][indz] = 1; return; }	
}
