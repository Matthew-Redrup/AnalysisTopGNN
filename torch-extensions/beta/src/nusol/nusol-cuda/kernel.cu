#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include "nusol.cu"
#include "operators.cu"
#include <cmath>

template <typename scalar_t>
__global__ void _ShapeKernel(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> inpt, 
        const unsigned int len_i, 
        const unsigned int len_k, 
        const unsigned int len_j, 
        const bool assign)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 

    if (idx >= len_i || idy >= len_k || idz >= len_j){ return; }
    if (assign){ out[idx][idy][idz] = inpt[(idx >= inpt.size(0)) ? 0 : idx][idy][idz]; return; }
    if (idy == idz){ out[idx][idy][idz] = inpt[0][0][idz]; }
}

template <typename scalar_t>
__global__ void _H_Base(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> beta2_b, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mass2_b, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> pmc_b, 

        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> beta2_mu, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mass2_mu, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> pmc_mu, 
 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> cos, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> mass2, 
        const unsigned int dim_i, const unsigned int dim_m)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_i || idy >= 3 || idz >= 3){ return; }    
    if (idy <= 1 && idz == 1){ return; }
    if (idy == 2 && (idz == 0 || idz == 2)){ return; }
    
    scalar_t   mass2_W   = mass2[(idx >= dim_m) ? 0 : idx][0]; 
    scalar_t   mass2_top = mass2[(idx >= dim_m) ? 0 : idx][1]; 
    scalar_t   mass2_nu  = mass2[(idx >= dim_m) ? 0 : idx][2];     
    mass2_W   *= mass2_W;  
    mass2_top *= mass2_top; 
    mass2_nu  *= mass2_nu;  

    scalar_t beta_mu   = sqrt(beta2_mu[idx][0]);
    scalar_t beta_b    = sqrt(beta2_b[idx][0]); 
    scalar_t sin       = sqrt(1 - cos[idx][0]*cos[idx][0]);   
 
    scalar_t x0p  = _x0(mass2_top, mass2_W, mass2_b[idx][0], pmc_b[idx][3]); 
    scalar_t x0   = _x0(mass2_W, mass2_mu[idx][0], mass2_nu, pmc_mu[idx][3]);

    scalar_t Sx   = _Sx(x0, beta2_mu[idx][0], beta_mu, pmc_mu[idx][3]); 
    scalar_t Sy   = _Sy(x0p, Sx, beta_b, cos[idx][0], sin); 

    scalar_t w    = _w(beta_mu , beta_b, cos[idx][0], sin); 
    scalar_t Om2  = _omega2(w, beta2_mu[idx][0]); 

    scalar_t coef = _coef(Sx, Sy, w, Om2); 
    scalar_t x1   = Sx - coef; 
    if (idy == 0 && idz == 2){ out[idx][idy][idz] = x1 - beta_mu * pmc_mu[idx][3]; return; }

    scalar_t y1   = Sy - w*coef; 
    if (idy == 1 && idz == 2){ out[idx][idy][idz] = y1; }

    scalar_t eps2 = _epsilon2(mass2_W, mass2_nu, beta2_mu[idx][0]); 
    scalar_t Z    = _Z(x1, Om2, Sy, Sx, w, mass2_W, x0, eps2); 
    if (idy == 2 && idz == 1){ out[idx][idy][idz] = Z; return; }
    if (idy == 1 && idz == 0){ out[idx][idy][idz] = w*Z/sqrt(Om2); return; }
    if (idy == 0 && idz == 0){ out[idx][idy][idz] = Z/sqrt(Om2); return; }
}

template <typename scalar_t>
__global__ void _Base_Matrix_H_Kernel(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> Ry, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> Rz, 

        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> RyT, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> RzT, 

        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> phi, 
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> theta, 
        const unsigned int dim_x)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_x || idy >= 6 || idz  >= 3){return;}
    if (idy < 3)
    { 
        _rz(Rz[idx][idy][idz], -phi[idx][0], idy, idz); 
        RzT[idx][idz][idy] = Rz[idx][idy][idz]; 
    }
    else
    {
        _pihalf(theta[idx][0]); 
        _ry(Ry[idx][idy%3][idz], theta[idx][0], idy%3, idz); 
        RyT[idx][idz][idy%3] = Ry[idx][idy%3][idz]; 
    }
}

template <typename scalar_t>
__global__ void _Base_Matrix_H_Kernel(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> Rx, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> RxT, 
        const unsigned int dim_x)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_x || idy >= 3 || idz  >= 3){return;}
    _rx(RxT[idx][idz][idy], -atan2(Rx[idx][2][0], Rx[idx][1][0]), idy, idz); 
}

template <typename scalar_t>
__global__ void _V0_deltaK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> X, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> dNu, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> met_xy, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> shape,  
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> H, 
        const unsigned int dim_i)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_i || idy >= 3 || idz >= 3){ return; } 

    dNu[idx][idy][idz] = met_xy[idx][idy][2 - idz] - H[idx][idy][idz]; 
    scalar_t dot_ji = 0; 
    for (unsigned int i(0); i < 3; ++i)
    {
        dot_ji += (met_xy[idx][i][2 - idz] - H[idx][i][idz])*shape[idx][idy][i]; 
    }
    X[idx][idz][idy] = dot_ji; 
} 

template <typename scalar_t>
__global__ void _DerivativeK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> X, 
        const unsigned int dim_i)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if ( idx >= dim_i || idy >= 3 || idz >= 3 ){ return; }
    out[idx][idy][idz] = 0; 
    if (idy == 2 || idz == 2){ return; }
    _pihalf(out[idx][idy][idz]);  
    _rz(out[idx][idy][idz], out[idx][idy][idz], idy, idz); 
}

template <typename scalar_t>
__global__ void _transSumK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> in, 
        const unsigned int dim_i)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if ( idx >= dim_i || idy >= 3 || idz >= 3 ){ return; }
    out[idx][idy][idz] = in[idx][idy][idz] + in[idx][idz][idy]; 
}

template <typename scalar_t>
__global__ void _SwapAB_K(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> DetA, 
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> DetB,
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> A, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> B,
        const unsigned int dim_i)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_i || idy >= 3 || idz >= 3){ return; }
    _swapAB(A[idx][idy][idz], B[idx][idy][idz], DetA[idx][0], DetB[idx][0]); 
} 

template <typename scalar_t>
__global__ void _imagineK(
        torch::PackedTensorAccessor64<scalar_t, 4, torch::RestrictPtrTraits> out,
        //torch::PackedTensorAccessor<bool, 2, torch::RestrictPtrTraits> msk,
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> eigs,   
        //const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> A, 
        //const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> B, 
        const unsigned int dim_eig, const unsigned int dim_i)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y%3; 
    const unsigned int idz = blockIdx.y/3; 
    const unsigned int id_eig = blockIdx.z; 
    if ( idx >= dim_i || idy >= 3 || id_eig >= dim_eig ){ return; }
     








}
