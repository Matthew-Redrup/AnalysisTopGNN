#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include "operators.cu"

template <typename scalar_t>
__global__ void _DotK(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> i, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> j, 
        const unsigned int dim_i, 
        const unsigned int dim_j)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    if (idx >= dim_i || idy >= dim_j){return;}
    dot_ij(i[idx][idy], j[idx][idy]); 
}

template <typename scalar_t>
__global__ void _DotK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out,
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> v1, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> v2, 
        const unsigned int dim_z, 
        const unsigned int dim_i1,
        const unsigned int dim_co, 
        const unsigned int grid)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idz >= dim_z || idy >= dim_co || idx >= grid){return;}
    const unsigned int id = idx/dim_i1;
    const unsigned int idx_ = idx%dim_i1; 
    dot_ij(out[idz][idx_][idy + id*dim_co], v1[idz][idx_][idy], v2[idz][idy][id]); 
}

template <typename scalar_t>
__global__ void _DotK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> v1, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> v2, 
        unsigned int dim_x, 
        unsigned int dim_y)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_x || idy >= dim_y || idz >= 3){return;}
    if (idz == 0){ dot_ij(out[idx][idy][idz], v1[idx][idy], v1[idx][idy]); return; }
    if (idz == 1){ dot_ij(out[idx][idy][idz], v2[idx][idy], v2[idx][idy]); return; }
    dot_ij(out[idx][idy][idz], v1[idx][idy], v2[idx][idy]); 
}

template <typename scalar_t>
__global__ void _SumK(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> pmc, 
        const unsigned int length, 
        const unsigned int len_j)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx >= length){ return; }
    for (unsigned int i(1); i < len_j; ++i)
    {
        sum(pmc[idx][0], pmc[idx][i]);  
    }
}

template <typename scalar_t>
__global__ void _SumK(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> out, 
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> tmp, 
        const unsigned int length, const unsigned int len_j)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    if (idx >= length || idy >= 3){ return; }
    for (unsigned int i(0); i < len_j; ++i)
    {
        sum(out[idx][idy], tmp[idx][i][idy]);  
        tmp[idx][i][idy] = 0; 
    }
}

template <typename scalar_t>
__global__ void _SumK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> mul, 
        const unsigned int dim_z,
        const unsigned int dim_x,  
        const unsigned int dim_y, 
        const unsigned int range)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idz >= dim_z || idy >= dim_y || idx >= dim_x){return;}
    
    for (unsigned int i(0); i < range; ++i)
    {
        sum(out[idz][idx][idy], mul[idz][idx][range*idy+i]);  
    }
}

template <typename scalar_t>
__global__ void _CosThetaK(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> inpt, 
        const unsigned int length)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx >= length){ return; }    
    costheta(inpt[idx][0], inpt[idx][0], inpt[idx][1], inpt[idx][2]); 
}

template <typename scalar_t>
__global__ void _SinThetaK(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> inpt, 
        const unsigned int length)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx >= length){ return; }    
    sintheta(inpt[idx][0], inpt[idx][0], inpt[idx][1], inpt[idx][2]); 
}

template <typename scalar_t>
__global__ void _RotK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> angle, 
        const unsigned int dim_x, const unsigned int dim_r)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_x || idy >= 3 || idz  >= 3){return;}
    if (dim_r == 0){_rx(out[idx][idy][idz], angle[idx][0], idy, idz); return;} 
    if (dim_r == 1){_ry(out[idx][idy][idz], angle[idx][0], idy, idz); return;}    
    if (dim_r == 2){_rz(out[idx][idy][idz], angle[idx][0], idy, idz); return;} 
}

template <typename scalar_t>
__global__ void _CoFactorK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> mtx, 
        const unsigned int dim_x, const unsigned int dim_y)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= dim_x || idy >= dim_y || idz  >= 3){return;}
    const unsigned int _y[12] = {1, 1, 2, 2, 0, 0, 2, 2, 0, 0, 1, 1}; 
    const unsigned int _z[12] = {1, 2, 1, 2, 0, 2, 0, 2, 0, 1, 0, 1}; 
    const unsigned int idy_ = idy*4; 
    const unsigned int idz_ = idz*4; 
    _det(
            out[idx][idy][idz], 
            mtx[idx][_y[idy_  ]][_z[idz_  ]], 
            mtx[idx][_y[idy_+1]][_z[idz_+1]], 
            mtx[idx][_y[idy_+2]][_z[idz_+2]], 
            mtx[idx][_y[idy_+3]][_z[idz_+3]]
    ); 
    if ((idy+idz)%2 == 1){ out[idx][idy][idz] *= -1; }
}

template <typename scalar_t>
__global__ void _DetDotK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> coeff, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> matrix, 
        const unsigned int len)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= len || idy >= 1 || idz >= 3){ return; } 
    out[idx][idy][idz]  = coeff[idx][idy][idz]*matrix[idx][idy][idz]; 
}

template <typename scalar_t>
__global__ void _DetSumK(
        torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> det, 
        const unsigned int len)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx >= len){ return; } 
    out[idx][0] = det[idx][0][0] + det[idx][0][1] + det[idx][0][2]; 
}

template <typename scalar_t>
__global__ void _InvK(
        torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> out, 
        const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> det, 
        const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> coef, 
        const unsigned int len)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int idy = blockIdx.y; 
    const unsigned int idz = blockIdx.z; 
    if (idx >= len || idy >= 3 || idz >= 3){ return; } 
    if (det[idx][0] == 0){ out[idx][idz][idy] = 0; return; }
    out[idx][idz][idy] = (1/det[idx][0]) * coef[idx][idy][idz]; 
}


