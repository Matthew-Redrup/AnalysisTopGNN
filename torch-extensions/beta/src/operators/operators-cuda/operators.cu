#include <hip/hip_runtime.h>

template <typename scalar_t>
__device__ __forceinline__ void dot_ij(scalar_t &p2, const scalar_t &p)
{
    p2 *= (p);     
}

template <typename scalar_t>
__device__ __forceinline__ void sum(scalar_t &p, const scalar_t &p_1)
{
    p += p_1; 
}
